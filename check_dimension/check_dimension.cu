#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void checkIndex(void){
  printf("threadIdx: (%d,%d,%d) blockIdx: (%d, %d, %d) blockDim: (%d, %d, %d) gridDim: (%d, %d, %d)\n",
	 threadIdx.x, threadIdx.y, threadIdx.z,
	 blockIdx.x, blockIdx.y, blockIdx.z,
	 blockDim.x, blockDim.y, blockDim.z,
	 gridDim.x, gridDim.y,gridDim.z
	 );
}

int main(int argc, char** argv){
  // データの要素の合計数
  int nElem = 6;

  // グリッドとブロック構造を定義
  //
  // thread < block < grid 
  //
  // dim3 はdim3宣言とよばれていて
  // dim3 grid(10,10)はgridの中に10x10のblockを起動する
  // dim3 block(8,8,8)はblock中に8x8x8のthreadを起動する
  
  dim3 block(3); // ブロック中に3 thread を起動する
  
  // 指定されたデータサイズとスレッド数からグリッド数の引数(ブロック数)を指定する
  // (6 + 3 - 1) /  3 =  2 
  dim3 grid((nElem + block.x - 1)/block.x);

  // グリッドサイズをホスト側から確認
  printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
  printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);

  // グリッドサイズをデバイス側から確認
  checkIndex<<<grid, block>>>();

  // デイバイスをリセット(これを記述しないと動かない)
  hipDeviceReset();
  return(0);
}
