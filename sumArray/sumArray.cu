#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

#define CHECK(call)                               \
{                                                 \
  const hipError_t error = call;                 \
  if(error != cudaSucess)                         \
  {                                               \
    printf("Error: %s:%d, ", __FIEL__, __LINE__); \
    printf("code:%d, reason: %s\n", error,        \
            hipGetErrorString(error));           \
    exit(1);                                      \
  }                                               \          
}
 
// ホストで計算した値とGPUで計算した値が同一かチェックする 
void checkResult(float *hostRef, float *gpuRef, cont int N){
  double epsilon = 1.0E-8;
  bool match = 1;
  for(int i = 0; i < N; i++){
    if(abs(hostRed[i] - gpuRef[i]) > epsilon){
      match = 0;
      printf("Arrays do not match!\n");
      printf("host %5.2f gpu %5.2f at current %d\n",hostRef[i], gpuRef[i],i);
      break;
    }
  }
  if(match){
    printf("Arrays match \n\n");
  }
}

// データ初期化
// 入力された配列要素を初期化する
void initialData(float *ip, int size){
  // 乱数シードの作成
  time_t t;
  srand((unsigned int)time(&t));
  for(int i = 0; i < size; i++){
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }
  return;
}

// ホスト側でベクトルの和を計算
void sumArraysOnHost(float *A, float *B, float *C, const int N){
  for(int idx = 0; idx < N; idx++){
    C[idx] = A[idx] + B[idx];
  }
}

// GPU 側でベクトルの和を計算
// カーネル関数の定義
// sumArraysOnHost に対して配列を操作するloopが存在しない!!
// 配列のループの代わりに複数のthread で計算する
__global__ void sumArrayOnGPU(float *A, float *B, float *c){
  // スレッドIDを割り当てる
  int i = threadId.x;
  C[i] = A[i] + B[i];
}


int main(int argc, char** argv){
  printf("%s Starting..\n", argv[0]);
  int dev = 0;         // デバイスのセットアップ
  hipSetDevice(dev);  // 0番目(1枚目) のデバイス(GPUカード)を利用する
  
  // ベクトルのデータサイズを設定する
  int nElem = 32;
  printf("Vector size %d\n", nElem);
  
  // ホスト側のメモリを確保する
  size_t nBytes = nElem * sizeof(float);
  float *h_A, *h_B, *hostRef, *gpuRef;
  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  hostRef = (float *)malloc(nBytes);
  gpuRef  = (float *)malloc(nBytes);
  
  
  // ホスト側で配列を初期化する
  initialData(h_A, nElem);
  initialData(h_B, nElem);
  memset(hostRef, 0, nBytes);
  memset(gpuRef, 0,  nBytes);
  
  // デバイス側のグローバルメモリを取得する
  float *d_A, *d_B, *d_C;
  // hipMalloc( void **devPtr, size_t size) の形式
  // 入力は void のダブルポインタ
  // 参考書ではここの表現が揺れている
  hipMalloc((void**)&d_A, nBytes);
  hipMalloc((void**)&d_B, nBytes);
  hipMalloc((void**)&d_C, nBytes);
  

  // ホストからデバイスへデータ転送
  // CPU -> GPU 
  hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice);
  
  // ホスト側でカーネルを呼び出す
  dim3 block(nElem);
  dim3 grid(1);
  
  sumArrayOnGPU <<grid, block>>(d_A, d_B, d_C, nElem);
  printf("Execution configure << %d, %d >>\n", grid.x, block.x);
  
  // カーネル関数の結果をホスト側にコピー
  hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
  
  // 結果をチェックするためにホスト側でベクトルの加算
  sumArraysOnHost(h_A, h_B, hostRef, nElem);
  
  // デバイスの結果をチェック
  checkResult(hostRef, gpuRef, nElem);
  
  // デバイスのメモリを開放
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  
  // ホストのメモリを開放
  free(h_A);
  free(h_B);
  free(hostRef);
  free(gpuRef);
  
  hipDeviceReset();
  return(0);
}
