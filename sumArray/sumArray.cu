
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

// ホスト側でベクトルの和を計算
void sumArraysOnHost(float *A, float *B, float *C, const int N){
  for(int idx = 0; idx < N; idx++){
    C[idx] = A[idx] + B[idx];
  }
}

//データ初期化
void initialData(float *ip, int size){
  // 乱数シードの作成
  time_t t;
  srand((unsigned int)time(&t));
  for(int i = 0; i < size; i++){
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }
  return;
}

int main(int argc, char** argv){
  int nElem = 1024;
  size_t nByte = nElem * sizeof(float);
  // ホスト側の変数にはプレフィックス "h_" を付与する 
  float *h_A, *h_B, *h_C;
  h_A = (float *)malloc(nByte);
  h_B = (float *)malloc(nByte);
  h_C = (float *)malloc(nByte);
  // 配列の要素を初期化
  initialData(h_A, nElem);
  initialData(h_B, nElem);

  // ホストで配列の和を計算
  sumArraysOnHost(h_A, h_B, h_C, nElem);

  // メモリを開放
  free(h_A);
  free(h_B);
  free(h_C);
  return(0);
}
