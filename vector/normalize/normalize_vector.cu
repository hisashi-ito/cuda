#include "hip/hip_runtime.h"
//
// 【normalize_vector】
//
//  概要: ベクトルの正規化関数サンプル
//        参考:
//        CUDA for Engineers: An Introduction to High-Performance Parallel Computing
//
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <cmath>
#include <iostream>
using namespace std;

// normalize vector
void normalize(thrust::device_vector<double> &v){
  double norm = sqrt(thrust::inner_product(v.begin(), v.end(), v.begin(), 0.0));
  using namespace thrust::placeholders;
  thrust::transform(v.begin(), v.end(), v.begin(), _1 /= norm);
}

int main(){
  thrust::device_vector<double> vec(2);
  vec[0] = 1.0;
  vec[1] = 2.0;
  // ベクトルの正規化
  normalize(vec);
  // 確認
  for(int i = 0; i < vec.size(); i++){
    cout << "vec[" << i << "] = " << vec[i] << endl;
  }
}
