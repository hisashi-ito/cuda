//
// 【coo2csr】
//
//  概要: cuSPARSE 行列変換サンプル
//        coo保存形式からcsr形式への変換する
//
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipsparse.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <vector>
using namespace std;

int main(){

  // Matrix
  // ^^^^^^
  // 3 0 0
  // 6 0 0
  // 0 2 1

  // COO 形式
  thrust::host_vector<double> h_values;
  thrust::host_vector<int> h_rows;
  thrust::host_vector<int> h_cols;
  
  // non-zero 成分を保存
  h_values.push_back(3.0);
  h_values.push_back(6.0);
  h_values.push_back(2.0);
  h_values.push_back(1.0);
  // 縦方向
  h_cols.push_back(0);
  h_cols.push_back(0);
  h_cols.push_back(1);
  h_cols.push_back(2);
  // 横方向
  h_rows.push_back(0);
  h_rows.push_back(1);
  h_rows.push_back(2);
  h_rows.push_back(2);
  
  // [デバイス側]
  // cuSPARSE のハンドルを作成
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  // non-zero 要素数
  int nnz = h_values.size();
  // 変換前の行列の行数(rows)
  int rsize = max_element(h_rows.begin(), h_rows.end());
  
  // デバイス側でCOO形式のデバイスメモリを取得
  // ただし、CSR形式への変換はh_values, h_cols は変更必要ない
  // h_rows だけが変更がなされる
  thrust::device_vector<double> d_values = h_values;
  thrust::device_vector<int> d_cols = h_cols;
  thrust::device_vector<int> d_rows = h_rows;
  thrust::device_vector<int> d_csr_rows(4);
  
  // 行列のディスクリプタを記述
  hipsparseMatDescr_t matDescr;
  hipsparseCreateMatDescr(&matDescr);
  hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);

  // COO -> CSR 形式へ変換(rowsだけ)
  hipsparseXcoo2csr(handle,thrust::raw_pointer_cast(&d_rows[0]),nnz,rsize,
		   thrust::raw_pointer_cast(&d_csr_rows[0]),HIPSPARSE_INDEX_BASE_ZERO);
  
  // 計算結果を確認
  for(int i =0; i< d_csr_rows.size(); i++){
    cout << d_csr_rows[i] << endl;
  }
  exit(1);
}
