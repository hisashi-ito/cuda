/*
 * PageRankViennaCL.cu
 *
 *  Created on: Jun 30, 2015
 *      Author: yongchao
 */

/*ViennaCL*/
#include "PageRank.h"
struct viennaclOptions {
	viennaclOptions() {
		/*default settings*/
		_gpuId = 0;
		_nrepeats = 10;
		_sparseMatFormat = CSR_FORMAT;
		_doublePrecision = false;
	}

	/*variables*/
	int _gpuId;
	int _nrepeats;
	int _sparseMatFormat; /*sparse matrix format*/
	string _graphFileName; /*adjacency matrix of the graph stored in sparse matrix*/
	bool _doublePrecision; /*use single precision*/
	vector<int> _gpus;

	void printUsage() {
		cerr << "PageRankCuda viennacl [options]" << endl << "Options: " << endl
				<< "\t-i <str> (sparse matrix file name)" << endl
				<< "\t-d <int> (use double-precision floating point, default="
				<< _doublePrecision << ")" << endl
				<< "\t-r <int> (number of repeated runs, default=" << _nrepeats
				<< ")" << endl << "\t-g <int> (GPU index to use, default="
				<< _gpuId << ")" << endl << endl;
	}
	bool parseArgs(int argc, char* argv[]) {
		int c;

		/*GPU information*/
		int count;
		hipDeviceProp_t prop;
		hipGetDeviceCount(&count);
		CudaCheckError();

#if defined(HAVE_SM_35)
		cerr << "Require GPUs with compute capability >= 3.5" << endl;
#else
		cerr << "Require GPUs with compute capability >= 3.0" << endl;
#endif
		/*check the compute capability of GPUs*/
		for (int i = 0; i < count; ++i) {
			hipGetDeviceProperties(&prop, i);
#if defined(HAVE_SM_35)
			if ((prop.major * 10 + prop.minor) >= 35) {
#else
			if ((prop.major * 10 + prop.minor) >= 30) {
#endif
				cerr << "GPU " << _gpus.size() << ": " << prop.name
						<< " (capability " << prop.major << "." << prop.minor
						<< ")" << endl;
				/*save the GPU*/
				_gpus.push_back(i);
			}
		}
		if (_gpus.size() == 0) {
			cerr << "No qualified CUDA-enabled GPU is available" << endl;
			return false;
		}
		cerr << "Number of qualified GPUs: " << _gpus.size() << endl;

		/*parse parameters*/
		while ((c = getopt(argc, argv, "i:d:r:g:\n")) != -1) {
			switch (c) {
			case 'i':
				_graphFileName = optarg;
				break;
			case 'g':
				_gpuId = atoi(optarg);
				break;
			case 'd':
				_doublePrecision = atoi(optarg) == 0 ? false : true;
				break;
			case 'r':
				_nrepeats = atoi(optarg);
				if (_nrepeats < 1) {
					_nrepeats = 1;
				}
				break;
			default:
				cerr << "Unknown command: " << optarg << endl;
				return false;
			}
		}
		/*check the file name*/
		if (_graphFileName.length() == 0) {
			cerr << "Graph must be given" << endl;
			return false;
		}

		/*check GPU ID*/
		if (_gpuId >= (int) _gpus.size()) {
			_gpuId = _gpus.size() - 1;
		}
		if (_gpuId < 0) {
			_gpuId = 0;
		}
		return true;
	}
};

int main_viennacl_pagerank(int argc, char* argv[]) {
	viennaclOptions options;

	/*parse parameters*/
	if (options.parseArgs(argc, argv) == false) {
		options.printUsage();
		return -1;
	}

	/*set the GPU device*/
	hipSetDevice(options._gpus[options._gpuId]);
	CudaCheckError();

	/*perform SpMV*/
	bool ret = false;
	switch (options._sparseMatFormat) {
	case CSR_FORMAT:
		if (options._doublePrecision) {
			/*using double precision*/
			ret = pageRankViennaCL<unsigned int, double>(
					options._graphFileName.c_str(), options._nrepeats);

		} else {
			/*using single precision*/
			ret = pageRankViennaCL<unsigned int, float>(
					options._graphFileName.c_str(), options._nrepeats);
		}
		break;
	}
	return ret ? 0 : -1;
}

