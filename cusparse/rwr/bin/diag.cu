//
// Name: diag
//
// File Name:   diag.cu (definition file)
// Header file: dig.h   (header file)
//
// 概要: 対角化を実行するクラス
//       現在は冪情報のみを扱う
//       (厳密にはさgoogle matrixを対角化する)
//
//       CUDA のポインタ管理には thrustを利用する
//
// 更新履歴:
//          2018.04.03 新規作成
//
#include 'diag.h'

// @constructor
// @param: coo_file   行列(COO形式)ファイル名
// @param: iteration  冪情報の繰り返し回数
// @param: aplha      google パラメータ
//
Diag::Diag(const string coo_file, int iteration, double alpha){
  // パラメータをインスタンス変数へ保存する
  this->iteration = iteration;
  this->alpha     = alpha;
  
  // [ホスト側]
  //  COO形式の行列を読み込む 
  load_matrix(coo_file, this->h_rows, this->h_columns, this->h_values);
  
  // デバイス側の準備
  // cuSPARSE のハンドルを作成
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  
  // non-zero 要素数
  this->nnz = this->h_values.size();
  this->row_size = max_element(h_rows.begin(), h_rows.end()) ;      // 変換前の行列の行数(rows)
  this->col_size = max_element(h_columns.begin(), h_columns.end()); // 変換前の行列の行数(colms)
  
  // デバイス側でCOO形式のデバイスメモリを取得
  // ただし、CSR形式への変換はh_values, h_cols は変更必要ない。h_rows だけが変更がなされる
  thrust::device_vector<double> d_csr_values = h_values;
  thrust::device_vector<int> d_csr_cols = h_cols;
  thrust::device_vector<int> d_rows = h_rows;
  thrust::device_vector<int> d_csr_rows(r_size+1);
  
  // 行列のディスクリプタを記述
  hipsparseMatDescr_t matDescr;
  hipsparseCreateMatDescr(&matDescr);
  hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);

  // COO -> CSR 形式へ変換(rowsだけ)
  
  hipsparseXcoo2csr(handle,thrust::raw_pointer_cast(&d_rows[0])
		   ,this->row_size, this->col_size, this->nnz, thrust::raw_pointer_cast(&d_csr_rows[0]),
		   HIPSPARSE_INDEX_BASE_ZERO);

  

}

// @destructor
Diag::~Diag(void){}

// @power_method
Diag::power_method(thrust::host_vector<double> &x, 
		   thrust::host_vector<double> &ret){
  // ベクトル情報をGPUへオフロード
  thrust::device_vector<double> x = x;
  thrust::device_vector<double> y(x.size());
  
  //cuSPARSE のハンドルを作成
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  hipsparseMatDescr_t matDescr;
  hipsparseCreateMatDescr(&matDescr);
  hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);
  
  double* _d_csr_values = thrust::raw_pointer_cast(&(d_csr_values[0]));
  double* _d_csr_cols   = thrust::raw_pointer_cast(&(d_csr_cols[0]));
  double* _d_csr_rows   = thrust::raw_pointer_cast(&(d_csr_rows[0]));
  double* _x            = thrust::raw_pointer_cast(&(x[0]));
  double* _y            = thrust::raw_pointer_cast(&(y[0]));
  double beta = 1.0 - this->alpha;
  
  for(int i = 0; i < this->iteration, i++){
    // y = alpha * Ax + beta * y
    hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		   );
  }
  

		 
  
}


// @load_matrix
// @bref:  COO形式の行列を読み込む。読み込み後は
//         rows, columns, values に保存される
// @param: file 行列(COO形式)ファイル名
// @param: rows, colums, values (COO行列のベクトル)
//
void Diag::load_matrix(const string file, thrust::host_vector<int> &rows, 
		  thrust::host_vector<int> &columns, thrust::host_vector<double> &values){
  string buff;
  ifstream ifs(file.c_str());
  if(ifs.fail()){
    cerr << "[error] 遷移行列のファイルの読み込みに失敗しました" << endl;
    exit(-1);
  }
  // 遷移行列の読み込み
  // row, colums でソートされていることを期待します!
  // フォーマット) row<TAB>column<TAB>value
  while(getline(ifs, buff)){
    vector<string> elems = split(buff, '\t');
    int row    = atoi(elems[0].c_str());
    int col    = atoi(elems[1].c_str());
    double val = (double)atof(elems[2].c_str());
    // vector へpushする
    rows.push_back(row);
    columns.push_back(col);
    values.push_back(val);
  }
}

// @split
//  文字列分割関数
// @breaf 指定のデリミネタを利用してstringの文字列を分割する
//        分割された文字列は vector<string> 文字列に格納される
// @param 文字列(string)
// @param 分割デリミネタ(char)
// @return 分割された文字列が vector<string>で返却される
//
vector<string> Diag::split(const string &s, char delim){
  vector<string> elems;
  stringstream ss(s);
  string item;
  while (getline(ss, item, delim)) {
    if (!item.empty()){
      elems.push_back(item);
    }
  }
  return elems;
}
