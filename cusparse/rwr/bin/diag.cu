#include "hip/hip_runtime.h"
//
// Name: diag
//
// File Name:   diag.cu (definition file)
// Header file: dig.h   (header file)
//
// 概要: 対角化を実行するクラス
//       対角化法はPOWER MTHOD のみを扱う
//       (厳密にはさgoogle matrixを対角化する)
//
//       CUDA のポインタ管理には thrustを利用する
//
// 更新履歴:
//          2018.04.03 新規作成
//
#include 'diag.h'

// @constructor
//  コンストラクタ
// @param: coo_file   行列(COO形式)ファイル名
// @param: iteration  冪情報の繰り返し回数
// @param: aplha      google パラメータ
//
Diag::Diag(const string coo_file, int iteration, double alpha){
  // パラメータをインスタンス変数へ保存する
  this->iteration = iteration;
  this->alpha     = alpha;
  
  // [ホスト側]
  //  COO形式の行列を読み込む 
  load_matrix(coo_file, this->h_rows, this->h_cols, this->h_vals);
  
  // デバイス側の準備
  // cuSPARSE のハンドルを作成
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  
  // non-zero 要素数
  this->nnz = this->h_vals.size();
  this->row_size = *max_element(h_rows.begin(), h_rows.end()) ; // 変換前の行列の行数(rows)
  this->col_size = *max_element(h_cols.begin(), h_cols.end());  // 変換前の行列の行数(colms)
  
  // デバイス側でCOO形式のデバイスメモリを取得
  // ただし、CSR形式への変換はh_vals, h_cols は変更必要ない。
  // h_rows だけが変更がなされる
  thrust::device_vector<int>    d_csr_cols = h_cols;
  thrust::device_vector<double> d_csr_vals = h_vals;
  thrust::device_vector<int> d_rows = h_rows;
  thrust::device_vector<int> d_csr_rows(this->row_size+1);
  
  // 行列のディスクリプタを記述
  hipsparseMatDescr_t matDescr;
  hipsparseCreateMatDescr(&matDescr);
  hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);

  // COO->CSR 形式へ変換(rowsだけ)
  hipsparseXcoo2csr(handle, thrust::raw_pointer_cast(&d_rows[0]),
		   this->nnz, this->row_size,
		   thrust::raw_pointer_cast(&d_csr_rows[0]),
		   HIPSPARSE_INDEX_BASE_ZERO);
}

// @destructor
//  デストラクタ
Diag::~Diag(void){
  // 行列要素等をfreeすべきだが、thrustを
  // 利用しているので自動で返却される
}

// @split
//  文字列分割関数
// @breaf 指定のデリミネタを利用してstringの文字列を分割する
//        分割された文字列は vector<string> 文字列に格納される
// @param 文字列(string)
// @param 分割デリミネタ(char)
// @return 分割された文字列が vector<string>で返却される
//
vector<string> Diag::split(const string &s, char delim){
  vector<string> elems;
  stringstream ss(s);
  string item;
  while (getline(ss, item, delim)) {
    if (!item.empty()){
      elems.push_back(item);
    }
  }
  return elems;
}

// @load_matrix
//  行列読み込み関数
// @breaf: COO形式の行列を読み込む。読み込み後はrows, cols, vals に保存される
// @param: file 行列(COO形式)ファイル名
// @param: rows, cols, vals (COO行列のベクトル)
//
void Diag::load_matrix(const string file, thrust::host_vector<int> &rows, 
		       thrust::host_vector<int> &cols, thrust::host_vector<double> &vals){
  string buff;
  ifstream ifs(file.c_str());
  if(ifs.fail()){
    cerr << "[error] 遷移行列のファイルの読み込みに失敗しました" << endl;
    exit(-1);
  }
  // 遷移行列の読み込み
  // row, colums でソートされていることを期待します!
  // フォーマット) row<TAB>column<TAB>value
  while(getline(ifs, buff)){
    vector<string> elems = split(buff, '\t');
    int row    = atoi(elems[0].c_str());
    int col    = atoi(elems[1].c_str());
    double val = (double)atof(elems[2].c_str());
    // vector へpushする
    rows.push_back(row);
    cols.push_back(col);
    vals.push_back(val);
  }
}

// @power_method
//  冪乗法
// @breaf: 対角化関数、実際にはG-matrix を対角化する
// @param: 
//
void Diag::power_method(thrust::host_vector<double> &x, 
			thrust::host_vector<double> &y){
  // ベクトル情報をGPUへオフロード
  thrust::device_vector<double> x = x;          
  thrust::device_vector<double> init_x(x.size());
  thrust::device_vector<double> y(x.size());
  
  // xをinit_x にdeep copyする
  thrust::copy(x.begin(), x.end(), init_x);
  // init_x をbeta 倍する
  const_multiplies(init_x, beta);
  
  // cuSPARSE のハンドルを作成(必要ないかもしれない) 
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  hipsparseMatDescr_t matDescr;
  hipsparseCreateMatDescr(&matDescr);
  hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);
  
  double* _x = thrust::raw_pointer_cast(&(x[0]));
  double* _y = thrust::raw_pointer_cast(&(y[0]));
  double* _d_csr_vals = thrust::raw_pointer_cast(&(d_csr_vals[0]));
  double* _d_csr_cols = thrust::raw_pointer_cast(&(d_csr_cols[0]));
  double* _d_csr_rows = thrust::raw_pointer_cast(&(d_csr_rows[0]));
  double beta  = 1.0 - this->alpha;
  double dummy = 0.0;
  
  for(int i = 0; i < this->iteration, i++){
    // y = α ∗ A ∗ x + (0 * y)
    hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		   this->row_size, this->col_size, this->nnz,
		   &this->alpha, matDescr,
		   _d_csr_vals, _d_csr_rows, _d_csr_cols,
		   _x, &dummy, _y);
    
    // デバイスポインタの変換
    x = thrust::raw_pointer_cast(&(_x[0]));
    y = thrust::raw_pointer_cast(&(_y[0]));
    // y += (β * init_x)
    thrust::transform(y.begin(), y.end(), init_x.begin(), y.begin(),thrust::plus<double>());
    // y をnormalizeする
    normalize(y);
    // y → x
    thrust::copy(y.begin(), y.end(), x);
    // デバイスポインタへ戻す
    _x = thrust::raw_pointer_cast(&(x[0]));
    _y = thrust::raw_pointer_cast(&(y[0]));
  }
  // デバイスから計算結果を返却
  thrust::copy_n(_y.begin(), _y.end(), y);
}

// @normalize
//  行列の正規化関数
// @breaf: ベクトルを正規化する
// @param: ベクトル
//
void Diag::normalize(thrust::device_vector<double> &v){
  double norm = sqrt(thrust::inner_product(v.begin(), v.end(), v.begin(), 0.0));
  using namespace thrust::placeholders;
  thrust::transform(v.begin(), v.end(), v.begin(), _1 /= norm);
}

// @const_multiplies
//  ベクトルの低数倍
// @bread: 行列を低数倍する
// @param: ベクトル
//
void Diag::const_multiplies(thrust::device_vector<double> &v, double alpha){
  using namespace thrust::placeholders;
  thrust::transform(v.begin(), v.end(), v.begin(), _1 *= alpha);
}
