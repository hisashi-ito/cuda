#include "hip/hip_runtime.h"
//
// Name: diag
//
// File Name:   diag.cu (definition file)
// Header file: dig.h   (header file)
//
// 概要: 対角化を実行するクラス
//       対角化法はPOWER MTHOD のみを扱う
//       (厳密にはさgoogle matrixを対角化する)
//
//       CUDA のポインタ管理には thrustを利用する
//
// 更新履歴:
//          2018.04.03 新規作成
//
#include "diag.h"

// @constructor
//  コンストラクタ
// @param: coo_file   行列(COO形式)ファイル名
// @param: iteration  冪情報の繰り返し回数
// @param: aplha      google パラメータ
//
Diag::Diag(const string coo_file, int iteration, double alpha){
  // パラメータをインスタンス変数へ保存する
  this->iteration = iteration;
  this->alpha = alpha;
  this->util  = new Util();
  
  // [ホスト側]
  //  COO形式の行列を読み込む 
  load_matrix(coo_file, this->h_rows, this->h_cols, this->h_vals);
  
  // デバイス側の準備
  // cuSPARSE のハンドルを作成
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  
  // non-zero 要素数
  this->nnz = this->h_vals.size();
  this->row_size = *max_element(h_rows.begin(), h_rows.end()) ; // 変換前の行列の行数(rows)
  this->col_size = *max_element(h_cols.begin(), h_cols.end());  // 変換前の行列の行数(colms)
  
  // デバイス側でCOO形式のデバイスメモリを取得
  // ただし、CSR形式への変換はh_vals, h_cols は変更必要ない。
  // h_rows だけが変更がなされる
  this->d_csr_cols = h_cols;
  this->d_csr_vals = h_vals;
  this->d_rows     = h_rows;
  this->d_csr_rows = thrust::host_vector<int>(this->row_size+1);
  
  // 行列のディスクリプタを記述
  hipsparseMatDescr_t matDescr;
  hipsparseCreateMatDescr(&matDescr);
  hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);

  // COO->CSR 形式へ変換(rowsだけ)
  hipsparseXcoo2csr(handle, thrust::raw_pointer_cast(&d_rows[0]),
		   this->nnz, this->row_size,
		   thrust::raw_pointer_cast(&d_csr_rows[0]),
		   HIPSPARSE_INDEX_BASE_ZERO);
}

// @destructor
//  デストラクタ
Diag::~Diag(void){
  // 行列要素等をfreeすべきだが、thrustを
  // 利用しているので自動で返却される
}

// @load_matrix
//  行列読み込み関数
// @breaf: COO形式の行列を読み込む。読み込み後はrows, cols, vals に保存される
// @param: file 行列(COO形式)ファイル名
// @param: rows, cols, vals (COO行列のベクトル)
//
void Diag::load_matrix(const string file, thrust::host_vector<int> &rows, 
		       thrust::host_vector<int> &cols, thrust::host_vector<double> &vals){
  string buff;
  ifstream ifs(file.c_str());
  if(ifs.fail()){
    cerr << "[error] 遷移行列のファイルの読み込みに失敗しました" << endl;
    exit(-1);
  }
  // 遷移行列の読み込み
  // row, colums でソートされていることを期待します!
  // フォーマット) row<TAB>column<TAB>value
  while(getline(ifs, buff)){
    vector<string> elems = this->util->split(buff, '\t');
    int row    = atoi(elems[0].c_str());
    int col    = atoi(elems[1].c_str());
    double val = (double)atof(elems[2].c_str());
    // vector へpushする
    rows.push_back(row);
    cols.push_back(col);
    vals.push_back(val);
  }
}

// @power_method
//  冪乗法
// @breaf: 対角化関数、実際にはG-matrix を対角化する
// @param: 推薦元の初期ベクトル
// @param: 計算結果ベクトル(固有Vector) 
//
void Diag::power_method(thrust::host_vector<double> &h_x, 
			thrust::host_vector<double> &h_y){
  
  //
  thrust::host_vector<double> _h_x = h_x;
  // ベクトル情報をGPUへオフロード
  thrust::device_vector<double> d_x = _h_x;
  /*
  thrust::device_vector<double> d_y(h_x.size());
  thrust::device_vector<double> d_init_x(h_x.size());
  
  // d_x → d_init_x
  thrust::copy(d_x.begin(), d_x.end(), d_init_x.begin());
  // d_init_x → beta(1-alpha) * d_init_x
  double beta = 1.0 - this->alpha;
  const_multiplies(d_init_x, beta);
  
  // cuSPARSE のハンドルを作成(必要ないかもしれない) 
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  hipsparseMatDescr_t matDescr;
  hipsparseCreateMatDescr(&matDescr);
  hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);
  
  double* _d_x = thrust::raw_pointer_cast(&(d_x[0]));
  double* _d_y = thrust::raw_pointer_cast(&(d_y[0]));
  double* _d_csr_vals = thrust::raw_pointer_cast(&(this->d_csr_vals[0]));
  int* _d_csr_cols = thrust::raw_pointer_cast(&(this->d_csr_cols[0]));
  int* _d_csr_rows = thrust::raw_pointer_cast(&(this->d_csr_rows[0]));
  double dummy = 0.0;
  
  for(int i = 0; i < this->iteration; i++){
    // y = α ∗ A ∗ x + (0 * y)
    hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		   this->row_size, this->col_size, this->nnz,
		   &this->alpha, matDescr,
		   _d_csr_vals, _d_csr_rows, _d_csr_cols,
		   _d_x, &dummy, _d_y);
    
    // raw ポインタからデバイスポインタへ変換
    &d_x[0] = thrust::device_pointer_cast(&(_d_x[0]));
    &d_y[0] = thrust::device_pointer_cast(&(_d_y[0]));
    // y += (β * init_x)
    thrust::transform(d_y.begin(), d_y.end(), d_init_x.begin(), d_y.begin(),thrust::plus<double>());
    // y をnormalizeする
    normalize(d_y);
    // y → x
    thrust::copy(d_y.begin(), d_y.end(), d_x.begin());
    // デバイスポインタをraw ポインタへ変換
    _d_x = thrust::raw_pointer_cast(&(d_x[0]));
    _d_y = thrust::raw_pointer_cast(&(d_y[0]));
  }
  
  // デバイスから計算結果を返却
  thrust::copy(d_y.begin(), d_y.end(), h_y.begin());
  */
}

// @normalize
//  行列の正規化関数
// @breaf: ベクトルを正規化する
// @param: ベクトル
//
void Diag::normalize(thrust::device_vector<double> &v){
  double norm = sqrt(thrust::inner_product(v.begin(), v.end(), v.begin(), 0.0));
  using namespace thrust::placeholders;
  thrust::transform(v.begin(), v.end(), v.begin(), _1 /= norm);
}

// @const_multiplies
//  ベクトルの低数倍
// @bread: 行列を低数倍する
// @param: ベクトル
//
void Diag::const_multiplies(thrust::device_vector<double> &v, double alpha){
  using namespace thrust::placeholders;
  thrust::transform(v.begin(), v.end(), v.begin(), _1 *= alpha);
}


#ifdef _DEBUG_
int main(){
  thrust::host_vector<double> vec(3);
  thrust::host_vector<double> ret(3);
  // 入力Vectorの初期化
  vec[0] = 0.01;
  vec[1] = 0.02;
  vec[2] = 0.01;
  ret[0] = 0.0;
  ret[1] = 0.0;
  ret[2] = 0.0;
  Diag *diag = new Diag("../data/matrix.tsv", 5, 0.85);
  diag->power_method(vec, ret);
  // 計算結果を表示
  for(int i = 0; i < 3; i++){
    cout << "ret[" << i << "]= " << ret[i] <<endl;
  }
  exit(0);
}
#endif /*_DEBUG_*/
