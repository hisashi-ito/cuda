//
// 【rwr】
// 
//  概要: Random Walk with Restart(RWR) の cuSPARSE版 実装
//        本コマンドは COO形式の遷移行列と初期ベクトルファイルを
//        読み込み初期ベクトルに対応した推薦結果を作成する
//        本コマンドは cuSPARSE ライブラリを利用する
//
//  usage: rwr -i <遷移行列> (COO形式)
//             -v <初期ベクトルファイル>
//             -a <rwr のalpha パラメータ> 
//             -o <出力>
//  更新履歴:
//           2018.04.02 新規作成
//
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <unistd.h>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include <sstream>
#include <cstdlib>
#include <typeinfo>
#include <iomanip>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

using namespace std;

// @split
//  文字列分割関数
// @breaf 指定のデリミネタを利用してstringの文字列を分割する
//        分割された文字列は vector<string> 文字列に格納される
// @param 文字列(string)
// @param 分割デリミネタ(char)
// @return 分割された文字列が vector<string>で返却される
//
vector<string> split(const string &s, char delim){
  vector<string> elems;
  stringstream ss(s);
  string item;
  while (getline(ss, item, delim)) {
    if (!item.empty()){
      elems.push_back(item);
    }
  }
  return elems;
}

// @load_maxtrix
// @breaf  COO形式の遷移行列を読み込み以下の配列要素(vector)を返却
//         rows, columns, values の３つ。
// @params 遷移行列のファイル名(string)
//         行配列(rows)
//         列配列(columns)
//         行列要素の値(values)
// @return void
//
void load_matrix(string file,
		 vector<int> &rows,
                 vector<int> &columns,
                 vector<double> &values){
  string buff;
  ifstream ifs(file.c_str());
  if(ifs.fail()){
    cerr << "[error] 遷移行列のファイルの読み込みに失敗しました" << endl;
    exit(-1);
  }
  // 遷移行列の読み込み
  // フォーマット) row<TAB>column<TAB>value
  while(getline(ifs, buff)){
    vector<string> elems = split(buff, '\t');
    int row    = atoi(elems[0].c_str());
    int col    = atoi(elems[1].c_str());
    double val = (double)atof(elems[2].c_str());
    // vector へpushする
    rows.push_back(row);
    columns.push_back(col);
    values.push_back(val);
  }
}

// @load_vec
// @breaf 推薦元となる初期ベクトルを読み込む
// @params ファイル名<string>
//         初期ベクトル(vec)
// @return void
//   
void load_vecs(string file, vector< vector<double> > &vecs){
  string buff;
  vector<double> tmp;
  ifstream ifs(file.c_str());
  if(ifs.fail()){
    cerr << "[error] 初期ベクトルファイルの読み込みに失敗しました" << endl;
    exit(-1);
  }
  
  // 初期ベクトルファイルの読み込み
  // フォーマット) 数値1<SP>数値2...
  while(getline(ifs, buff)){
    vector<string> elems = split(buff, ' ');
    for(int i = 0; i < elems.size(); i++){
      tmp.push_back((double)atof(elems[i].c_str()));
    }
    vecs.push_back(tmp);
    tmp.clear();
  }
}

// @main
// メイン関数
// @breaf Random Walk with Restart(RWR) の cuSPARSE版 実装のメイン関数
// @param  argc, *argv[]
// @return 正常終了時 0
//         不正終了時 負数
//
int main(int argc, char *argv[]){
  // 引数の処理
  int opt = 0;
  string trans_mat; // 遷移行列ファイル
  string init_vec;  // 初期べベクトルファイル
  string alpha;     // alpha パラメータ ・・・ google matrix parameter
  string output;    // 出力ファイル
  
  // COO 形式で遷移行列を表現する(ホスト側)
  vector<int> h_rows;
  vector<int> h_columns;
  vector<double> h_values;
  
  // 初期ベクトル
  vector< vector<double> > h_vecs;
  
  // 引数parse
  while((opt = getopt(argc, argv,"i:v:o:a:")) != -1){
    switch(opt){
    case 'i':
      trans_mat = optarg;
      break;
    case 'v':
      init_vec = optarg;
      break;
    case 'a':
      alpha = optarg;
      break;
    case 'o':
      output = optarg;
      break;
    case ':':  // no value applied
    case '?':  // invalid option
      exit(1);
    }
  }
  // 引数不正の場合,usage を出して終了
  if(trans_mat.empty() || init_vec.empty() || alpha.empty() || output.empty()){
    cerr << "[error] usage: rwr -i <trans_mat> -v <init_vec> -a <alpha>  -o <output>" << endl;
    exit(-1);
  }
  
  // [ホスト処理]
  //  COO形式の行列要素を読み込む
  load_matrix(trans_mat.c_str(), h_rows, h_columns,h_values);
  //  初期ベクトルを読み込む
  load_vecs(init_vec.c_str(), h_vecs);
  
  // [デバイス側]
  // cuSPARSE のハンドルを作成
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  
  // デバイス側でCOO形式のデバイスメモリを取得
  thrust::device_vector<int> d_rows(h_rows.size());
  thrust::device_vector<int> d_columns(h_columns.size());
  thrust::device_vector<double> d_values(h_values.size());
  
  // 行列Aのディスクリプタを記述
  hipsparseMatDescr_t matDescr;
  hipsparseCreateMatDescr(&matDescr);
  hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO);
  
  // COO形式の行列データをGPUへオフロード
  // ホストベクトルをデバイスに転送
  thrust::copy_n(h_rows, h_rows.size(), d_rows.begin());
  thrust::copy_n(h_columns, h_columns.size(), d_columns.begin());
  thrust::copy_n(h_values, h_values.size(), d_values.begin());
  
  // CSR形式のベクトルを取得するためにデバイスメモリを確保
  // non-zero element
  int nnz    = h_values.size();
  int n_rows = max_element(h_rows.begin(), h_rows.end()); 
  thrust::device_vector<int> d_csr_valus(nnz);
  thrust::device_vector<int> d_csr_columns(nnz);
  thrust::device_vector<int> d_csr_rows(n_rows);
  // COO形式からCSR形式へ変換
  //cusparseDcoo2csr(handle, d_rows, nnz, 
  
  exit(0);
}
