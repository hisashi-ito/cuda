
#include <hip/hip_runtime.h>
#include <stdio.h>

// カーネル関数
__global__ void helloFromGPU(){
  if(threadIdx.x == 5){
    printf("Hello World form GPU! thread %d\n",threadIdx.x);
  }
}

int main(int argc, char **argv){
  printf("Hello World from CPU!\n");
  // カーネル関数の呼び出し
  helloFromGPU<<<1, 10>>>();
  hipDeviceReset();
  return 0;
}
